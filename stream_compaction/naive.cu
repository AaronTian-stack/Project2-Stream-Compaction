#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#include <cassert>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void naive_scan_kernel(int n, int offset, int* d_in, int* d_out)
        {
            int index = threadIdx.x + blockIdx.x * blockDim.x;
            if (index >= n)
            {
                return;
            }

            if (index >= offset)
            {
				d_out[index] = d_in[index - offset] + d_in[index];
            }
            else
            {
				d_out[index] = d_in[index];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            assert(n > 0);

            constexpr auto threads = 128;
            dim3 blockSize((n + threads - 1) / threads);

            int* d_in_data;
        	int* d_out_data;
			hipMalloc(&d_in_data, n * sizeof(int));
			hipMalloc(&d_out_data, n * sizeof(int));

            hipMemcpy(d_in_data, idata, n * sizeof(int), hipMemcpyDefault);

            timer().startGpuTimer();

            for (int i = 1; i < n; i *= 2)
            {
                naive_scan_kernel<<<blockSize, threads>>>(n, i, d_in_data, d_out_data);
                std::swap(d_in_data, d_out_data);
            }

            timer().endGpuTimer();

            // Exclusive
            hipMemcpy(odata + 1, d_in_data, (n - 1) * sizeof(int), hipMemcpyDefault);

            hipFree(d_in_data);
            hipFree(d_out_data);
        }
    }
}
